#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void Add(float *A, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
	   A[i] = A[i] + A[i];
}

void test_bandwidth()
{
	hipEvent_t* ticktock;
	ticktock = (hipEvent_t*)malloc(2*sizeof(hipEvent_t));
	hipEventCreate(&ticktock[0]);
	hipEventCreate(&ticktock[1]);
	int iter = 10;
	printf("The bandwidth should stay be about the same each time:\n");
	for(int i = 128; i < 4096;i+=256 )
	{
		float time = 0.0f;
		float *gpu_data;
		int size = i*(1024*1024/4);
		size_t bytes = size*sizeof(float);
		hipMalloc((void**)&gpu_data, bytes);
		hipMemset(gpu_data,0,bytes);

		int block_size = (size/512) + 1;
		hipEventRecord(ticktock[0], 0);
		//run multiple iterations to saturate the GPU
		for(int j = 0; j < iter; j++)
			//transposeNaive<<<block_size,512>>>(gpu_data, size);
			Add<<<block_size,512>>>(gpu_data, size);


		hipEventRecord(ticktock[1], 0);
		hipEventSynchronize(ticktock[1]);
		hipEventElapsedTime(&time, ticktock[0], ticktock[1]);

		time = time/1000.0f; //seconds

		float GB = ((iter*i)/1024.0f);

		printf("Data size: %f GB; Bandwidth: %f GB/s\n",i/1024.0f,GB/time);

		hipFree(gpu_data);

		hipDeviceSynchronize();

	}


}



int main(int argc, char *argv[])
{
	test_bandwidth();
}