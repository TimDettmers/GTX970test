#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void Add(float *A, int size)
{
  const unsigned int numThreads = blockDim.x * gridDim.x;
  const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

  for (unsigned int i = idx;i < size; i += numThreads)
	   A[i] = A[i]+ A[i];
}

void test_bandwidth()
{
	hipEvent_t* ticktock;
	ticktock = (hipEvent_t*)malloc(2*sizeof(hipEvent_t));
	hipEventCreate(&ticktock[0]);
	hipEventCreate(&ticktock[1]);
	int iter = 10;
	printf("The bandwidth should stay be about the same each time:\n");
	size_t free = 0, total = 0;  
	hipMemGetInfo(&free,&total);
	double used_memory_in_MB = (total- free)/1024./1024.;
	printf("Memory used by OS %f GB\n",used_memory_in_MB/1024.);
	for(int i = 128; i < 4096 - used_memory_in_MB;i+=256 )
	{
		float time = 0.0f;
		float *gpu_data;
		int size = i*(1024*1024/4);
		size_t bytes = size*sizeof(float);
		hipMalloc((void**)&gpu_data, bytes);
		hipMemset(gpu_data,0,bytes);
		
		hipEventRecord(ticktock[0], 0);

		//run multiple iterations to saturate the GPU		
		for(int j = 0; j < iter; j++)
			Add<<<512,512>>>(gpu_data, size);
		

		hipEventRecord(ticktock[1], 0);
		hipEventSynchronize(ticktock[1]);
		hipEventElapsedTime(&time, ticktock[0], ticktock[1]);
		hipDeviceSynchronize();

		time = time/1000.0f; //seconds

		float GB = ((iter*i)/1024.0f); 

		printf("Data size: %f GB; Bandwidth: %f GB/s\n",i/1024.0f, GB/time);

		hipFree(gpu_data);


	}


}



int main(int argc, char *argv[])
{
	test_bandwidth();
	return 0;
}