// Copyright 2012 NVIDIA Corporation
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{

  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }

  return result;
}

const int TILE_DIM = 32;
const int BLOCK_ROWS = 8;
const int NUM_REPS = 10;

// Check errors and print GB/s
void postprocess(const float *ref, const float *res, int n, float ms)
{
  bool passed = true;
  for (int i = 0; i < n; i++)
    if (res[i] != ref[i]) {
      printf("%d %f %f\n", i, res[i], ref[i]);
      printf("%25s\n", "*** FAILED ***");
      passed = false;
      break;
    }
  if (passed)
    printf("%20.2f\n", 2 * n * sizeof(float) * 1e-6 * NUM_REPS / ms );
}

// simple copy kernel
// Used as reference case representing best effective bandwidth.
__global__ void copy(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[(y+j)*width + x] = idata[(y+j)*width + x];
}

// copy kernel using shared memory
// Also used as reference case, demonstrating effect of using shared memory.
__global__ void copySharedMem(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM * TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[(threadIdx.y+j)*TILE_DIM + threadIdx.x];
}

// naive transpose
// Simplest transpose; doesn't use shared memory.
// Global memory reads are coalesced but writes are not.
__global__ void transposeNaive(float *odata, const float *idata)
{
  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j+= BLOCK_ROWS)
    odata[x*width + (y+j)] = idata[(y+j)*width + x];
}

// coalesced transpose
// Uses shared memory to achieve coalesing in both reads and writes
// Tile width == #banks causes shared memory bank conflicts.
__global__ void transposeCoalesced(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}


// No bank-conflict transpose
// Same as transposeCoalesced except the first tile dimension is padded
// to avoid shared memory bank conflicts.
__global__ void transposeNoBankConflicts(float *odata, const float *idata)
{
  __shared__ float tile[TILE_DIM][TILE_DIM+1];

  int x = blockIdx.x * TILE_DIM + threadIdx.x;
  int y = blockIdx.y * TILE_DIM + threadIdx.y;
  int width = gridDim.x * TILE_DIM;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     tile[threadIdx.y+j][threadIdx.x] = idata[(y+j)*width + x];

  __syncthreads();

  x = blockIdx.y * TILE_DIM + threadIdx.x;  // transpose block offset
  y = blockIdx.x * TILE_DIM + threadIdx.y;

  for (int j = 0; j < TILE_DIM; j += BLOCK_ROWS)
     odata[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}


void bandwidth_test(int rows_cols, int argc, char **argv)
{
    const int nx = rows_cols;
    const int ny = rows_cols;
    const int mem_size = nx*ny*sizeof(float);

    dim3 dimGrid(nx/TILE_DIM, ny/TILE_DIM, 1);
    dim3 dimBlock(TILE_DIM, BLOCK_ROWS, 1);

    int devId = 0;
    if (argc > 1) devId = atoi(argv[1]);

    hipDeviceProp_t prop;
    checkCuda( hipGetDeviceProperties(&prop, devId));
    printf("\nDevice : %s\n", prop.name);
    printf("Matrix size: %d %d, Block size: %d %d, Tile size: %d %d\n",
           nx, ny, TILE_DIM, BLOCK_ROWS, TILE_DIM, TILE_DIM);
    printf("dimGrid: %d %d %d. dimBlock: %d %d %d\n",
           dimGrid.x, dimGrid.y, dimGrid.z, dimBlock.x, dimBlock.y, dimBlock.z);

    checkCuda( hipSetDevice(devId) );

    float *h_idata = (float*)malloc(mem_size);
    float *h_cdata = (float*)malloc(mem_size);
    float *h_tdata = (float*)malloc(mem_size);
    float *gold    = (float*)malloc(mem_size);

    float *d_idata, *d_tdata;
    checkCuda( hipMalloc(&d_idata, mem_size) );
    checkCuda( hipMalloc(&d_tdata, mem_size) );

    // check parameters and calculate execution configuration
    if (nx % TILE_DIM || ny % TILE_DIM) {
      printf("nx and ny must be a multiple of TILE_DIM\n");
      goto error_exit;
    }

    if (TILE_DIM % BLOCK_ROWS) {
      printf("TILE_DIM must be a multiple of BLOCK_ROWS\n");
      goto error_exit;
    }

    // host
    for (int j = 0; j < ny; j++)
      for (int i = 0; i < nx; i++)
        h_idata[j*nx + i] = j*nx + i;

    // correct result for error checking
    for (int j = 0; j < ny; j++)
      for (int i = 0; i < nx; i++)
        gold[j*nx + i] = h_idata[i*nx + j];

    // device
    checkCuda( hipMemcpy(d_idata, h_idata, mem_size, hipMemcpyHostToDevice) );

    // events for timing
    hipEvent_t startEvent, stopEvent;
    checkCuda( hipEventCreate(&startEvent) );
    checkCuda( hipEventCreate(&stopEvent) );
    float ms;

    // ------------
    // time kernels
    // ------------
    printf("%25s%25s\n", "Routine", "Bandwidth (GB/s)");

    // ----
    // copy
    // ----
    printf("%25s", "copy");
    checkCuda( hipMemset(d_tdata, 0, mem_size) );
    // warm up
    copy<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    checkCuda( hipEventRecord(startEvent, 0) );
    for (int i = 0; i < NUM_REPS; i++)
       copy<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    for (int i = 0; i < NUM_REPS; i++)
       copy<<<dimGrid, dimBlock>>>(d_idata, d_tdata);
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    checkCuda( hipMemcpy(h_cdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );
    postprocess(h_idata, h_cdata, nx*ny, ms);

    // -------------
    // copySharedMem
    // -------------
    printf("%25s", "shared memory copy");
    checkCuda( hipMemset(d_tdata, 0, mem_size) );
    // warm up
    copySharedMem<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    checkCuda( hipEventRecord(startEvent, 0) );
    for (int i = 0; i < NUM_REPS; i++)
       copySharedMem<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    for (int i = 0; i < NUM_REPS; i++)
       copySharedMem<<<dimGrid, dimBlock>>>(d_idata, d_tdata);
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    checkCuda( hipMemcpy(h_cdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );
    postprocess(h_idata, h_cdata, nx * ny, ms);

    // --------------
    // transposeNaive
    // --------------
    printf("%25s", "naive transpose");
    checkCuda( hipMemset(d_tdata, 0, mem_size) );
    // warmup
    transposeNaive<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    checkCuda( hipEventRecord(startEvent, 0) );
    for (int i = 0; i < NUM_REPS; i++)
       transposeNaive<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    for (int i = 0; i < NUM_REPS; i++)
       transposeNaive<<<dimGrid, dimBlock>>>(d_idata, d_tdata);
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    checkCuda( hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );
    postprocess(gold, h_tdata, nx * ny, ms);

    // ------------------
    // transposeCoalesced
    // ------------------
    printf("%25s", "coalesced transpose");
    checkCuda( hipMemset(d_tdata, 0, mem_size) );
    // warmup
    transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    checkCuda( hipEventRecord(startEvent, 0) );
    for (int i = 0; i < NUM_REPS; i++)
       transposeCoalesced<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    for (int i = 0; i < NUM_REPS; i++)
       transposeCoalesced<<<dimGrid, dimBlock>>>(d_idata, d_tdata);
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    checkCuda( hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );
    postprocess(gold, h_tdata, nx * ny, ms);

    // ------------------------
    // transposeNoBankConflicts
    // ------------------------
    printf("%25s", "conflict-free transpose");
    checkCuda( hipMemset(d_tdata, 0, mem_size) );
    // warmup
    transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    checkCuda( hipEventRecord(startEvent, 0) );
    for (int i = 0; i < NUM_REPS; i++)
       transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_tdata, d_idata);
    for (int i = 0; i < NUM_REPS; i++)
       transposeNoBankConflicts<<<dimGrid, dimBlock>>>(d_idata, d_tdata);
    checkCuda( hipEventRecord(stopEvent, 0) );
    checkCuda( hipEventSynchronize(stopEvent) );
    checkCuda( hipEventElapsedTime(&ms, startEvent, stopEvent) );
    checkCuda( hipMemcpy(h_tdata, d_tdata, mem_size, hipMemcpyDeviceToHost) );
    postprocess(gold, h_tdata, nx * ny, ms);

  error_exit:
    // cleanup
    checkCuda( hipEventDestroy(startEvent) );
    checkCuda( hipEventDestroy(stopEvent) );
    checkCuda( hipFree(d_tdata) );
    checkCuda( hipFree(d_idata) );
    free(h_idata);
    free(h_tdata);
    free(h_cdata);
    free(gold);

}

int main(int argc, char **argv)
{
  double size = 0.0;
  long long row_and_col_size = 20000;
  size_t free = 0, total = 0;  
  hipMemGetInfo(&free,&total);
  double used_memory_in_GB = (total- free)/1024./1024./1024.;
  while(size < 4.096 - used_memory_in_GB)
  {
    size = 2*(row_and_col_size*row_and_col_size*4)/1024./1024./1024.;
    printf("\n");
    printf("--------------------------------------------\n");
    printf("Allocated GPU memory size in GB: %f\n",size);
    printf("Used GPU memory by other applications in GB: %f\n",used_memory_in_GB);
    printf("--------------------------------------------\n");
    bandwidth_test(row_and_col_size, argc, argv);
    row_and_col_size+=256;
  }

}
